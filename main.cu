#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <numeric>

constexpr int POPULATION_SIZE = 30;
constexpr int COMBINATIONAL_CROSSING_SIZE = 104400;
constexpr int MAX_POPULATION_SIZE = 109890;
constexpr int CHROMOSOMES_SIZE = 183;
constexpr int MAX_POOL = 30;
constexpr int MAX_ITERATIONS = 3;

void initialize(int**& population, int* original, double*& dists, double*& sorted, int**& closest)
{
    srand(time(NULL));
    int x = 0;
    for (int i = 0; i < POPULATION_SIZE; i++)
    {
        for (int j = 0; j < CHROMOSOMES_SIZE; j++)
        {
            population[i][j] = x % 55;
            dists[i] += pow(original[j] - population[i][j], 2);
            x++;
        }
        dists[i] = sqrt(dists[i]);
        sorted[i] = dists[i];
    }

    std::sort(sorted, sorted + POPULATION_SIZE);

    for (int i = 0; i < MAX_POOL; i++)
    {
        for (int j = 0; j < CHROMOSOMES_SIZE; j++) closest[i][j] = population[std::distance(dists, std::find(dists, dists + POPULATION_SIZE, sorted[i]))][j];
    }
    printf("ISorted3 = %2.4f\tSum = %2.4f\n", sorted[0], std::accumulate(sorted, sorted + POPULATION_SIZE, 0.0));

}

void get_distances(int**& population, int* original, double*& distances, double*& sorted)
{
    for (int i = 0; i < MAX_POPULATION_SIZE; i++)
    {
        distances[i] = 0;
        for (int j = 0; j < CHROMOSOMES_SIZE; j++)
        {
            distances[i] += pow(original[j] - population[i][j], 2);
        }
        distances[i] = sqrt(distances[i]);
        sorted[i] = distances[i];
    }
}

void get_closest_distances(int**& population, int**& closest, double*& distances, double*& sorted)
{
    int iteration = 1;
    std::sort(sorted, sorted + MAX_POPULATION_SIZE);
    for (int i = 0; i < MAX_POOL; i++)
    {
        auto index = std::distance(distances, std::find(distances, distances + MAX_POPULATION_SIZE, sorted[i]));
        for (int j = 0; j < CHROMOSOMES_SIZE; j++)
        {
            closest[i][j] = population[index][j];
        }
    }
    // printf("Sorted = %2.4f\tSum = %2.4f\n", sorted[0], std::accumulate(sorted, sorted + MAX_POPULATION_SIZE, 0.0));
    iteration++;
}

void combinational_crossing(int**& population, int**& closest)
{
    auto inner_count = 0;

    for (int i = 0; i < MAX_POOL; i++)
    {
        auto N = 0;
        for (int j = i + 1; j < MAX_POOL; j++)
        {
            // printf("I: %d\tJ: %d\tCount: %d\tInnerCount: %d\n", i, j, N, inner_count);
            for (int p = 0; p < 60; p++)
            {
                for (int x = 0; x < CHROMOSOMES_SIZE; x++)
                {
                    population[inner_count + 0][x] = closest[i][x];
                    population[inner_count + 1][x] = closest[j][x];
                    population[inner_count + 2][x] = closest[i][x];
                    population[inner_count + 3][x] = closest[j][x];
                }

                population[inner_count + 0][3 * p + 0] = closest[j][3 * p + 3];
                population[inner_count + 0][3 * p + 1] = closest[j][3 * p + 4];
                population[inner_count + 0][3 * p + 2] = closest[j][3 * p + 5];

                population[inner_count + 1][3 * p + 0] = closest[i][3 * p + 3];
                population[inner_count + 1][3 * p + 1] = closest[i][3 * p + 4];
                population[inner_count + 1][3 * p + 2] = closest[i][3 * p + 5];

                population[inner_count + 2][3 * p + 3] = closest[j][3 * p + 0];
                population[inner_count + 2][3 * p + 4] = closest[j][3 * p + 1];
                population[inner_count + 2][3 * p + 5] = closest[j][3 * p + 2];

                population[inner_count + 3][3 * p + 3] = closest[i][3 * p + 0];
                population[inner_count + 3][3 * p + 4] = closest[i][3 * p + 1];
                population[inner_count + 3][3 * p + 5] = closest[i][3 * p + 2];

                inner_count += 4;
                N += 4;
            }
        }
    }
}

void mutate(int**& population, int**& closest, int offset)
{
    auto x = offset;
    for (int i = 0; i < MAX_POOL; i++)
    {
        for (int j = 0; j < CHROMOSOMES_SIZE; j++)
        {
            for (int k = 0; k < 183; k++) population[x][k] = closest[i][k];
            population[x][j] = j % 55;
            x++;
        }
    }
}

bool check(int* original, int**& population)
{
    for (int i = 0; i < MAX_POPULATION_SIZE; i++)
    {
        for (int j = 0; j < CHROMOSOMES_SIZE; j++)
        {
            if (population[i][j] != original[j]) break;
            if (j == CHROMOSOMES_SIZE - 1)
            {
                printf("Match Found at population %d\n", i); return true;
            }
        }
    }
    return false;
}

__device__ double get_euclidean_distance(int* sample, int* original)
{
    auto tid = threadIdx.x;
    __shared__ int s_dist[256];
    if (tid < 183)
    {
        s_dist[tid] = (original[tid] - sample[tid]);
        s_dist[tid] = s_dist[tid] * s_dist[tid];
    }
    __syncthreads();
    for (int s = 128; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_dist[tid] += s_dist[tid + s];
        }
        __syncthreads();
    }
    return s_dist[0];
}

__global__ void get_distances_gpu(int** population, int* original, double* dist)
{
    auto x = sqrt(get_euclidean_distance(population[blockIdx.x], original));
    if (blockIdx.x < 10 && threadIdx.x == 0) printf("Distance is %2.4f\n", x);
    dist[blockIdx.x] = x;
}

__global__ void combinational_crossing_gpu(int** population, int** closest)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    int offset = 0;
    for (int I = 0; I < i; I++)
        offset += 240 * (blockDim.x - (i + 1));

    if (j > i && i == 0)
    {
        printf("Thread: %d\t%d + %d\n", j, offset, 240 * (j - i - 1));
        offset += 240 * (j - i - 1);
        for (int p = 0; p < 60; p++)
        {
            for (int x = 0; x < CHROMOSOMES_SIZE; x++)
            {
                population[offset + 0][x] = closest[i][x];
                population[offset + 1][x] = closest[j][x];
                population[offset + 2][x] = closest[i][x];
                population[offset + 3][x] = closest[j][x];
            }

            population[offset + 0][3 * p + 0] = closest[j][3 * p + 3];
            population[offset + 0][3 * p + 1] = closest[j][3 * p + 4];
            population[offset + 0][3 * p + 2] = closest[j][3 * p + 5];

            population[offset + 1][3 * p + 0] = closest[i][3 * p + 3];
            population[offset + 1][3 * p + 1] = closest[i][3 * p + 4];
            population[offset + 1][3 * p + 2] = closest[i][3 * p + 5];

            population[offset + 2][3 * p + 3] = closest[j][3 * p + 0];
            population[offset + 2][3 * p + 4] = closest[j][3 * p + 1];
            population[offset + 2][3 * p + 5] = closest[j][3 * p + 2];

            population[offset + 3][3 * p + 3] = closest[i][3 * p + 0];
            population[offset + 3][3 * p + 4] = closest[i][3 * p + 1];
            population[offset + 3][3 * p + 5] = closest[i][3 * p + 2];

            offset += 4;
        }
    }
}

int main()
{
    srand(time(NULL));

    //Host Initializations
    double* distances = new double[MAX_POPULATION_SIZE];
    double* sorted = new double[MAX_POPULATION_SIZE];
    int** population = new int* [MAX_POPULATION_SIZE];
    int** closest = new int* [MAX_POOL];

    int iterations = 0;
    for (int i = 0; i < MAX_POPULATION_SIZE; i++) population[i] = new int[CHROMOSOMES_SIZE];
    for (int i = 0; i < MAX_POOL; i++) closest[i] = new int[CHROMOSOMES_SIZE];

    int original[] = { 1, 0, 1, 2, 10, 1, 4, 5, 6, 11, 3, 7, 8, 12, 0, 4, 6, 8, 13, 1, 5, 7, 14, 0,
           2, 3, 15, 0, 4, 8, 9, 16, 1, 6, 8, 9, 17, 0, 4, 6, 9, 18, 0, 13, 9, 19, 0, 10,
           3, 20, 0, 29, 47, 21, 0, 17, 4, 22, 4, 9, 23, 0, 30, 12, 24, 4, 39, 25, 0, 49, 14, 26,
           0, 25, 4, 9, 27, 2, 21, 50, 28, 2, 43, 37, 29, 0, 11, 20, 23, 30, 4, 2, 31, 3, 39, 19,
           32, 0, 47, 38, 33, 0, 16, 25, 41, 34, 0, 26, 47, 35, 0, 27, 18, 39, 36, 0, 52, 54, 37, 2,
           46, 15, 38, 0, 3, 31, 39, 2, 16, 22, 40, 4, 6, 41, 2, 35, 50, 42, 1, 2, 3, 43, 4, 0,
           44, 4, 8, 45, 0, 32, 14, 46, 4, 1, 47, 0, 40, 39, 48, 0, 34, 40, 49, 0, 25, 1, 50, 0,
           42, 44, 51, 0, 14, 45, 52, 0, 28, 44, 53, 2, 27, 36, 54 };

    initialize(population, original, distances, sorted, closest);
    combinational_crossing(population, closest);
    mutate(population, closest, COMBINATIONAL_CROSSING_SIZE);

    //Device Initializations
    double* d_distances;
    double* d_sorted;
    int* d_original;
    int** d_population;
    int** d_closest;

    hipMalloc((void**)&d_distances, MAX_POPULATION_SIZE * sizeof(double*));
    hipMalloc((void**)&d_sorted, MAX_POPULATION_SIZE * sizeof(double*));
    hipMalloc((void**)&d_original, CHROMOSOMES_SIZE * sizeof(int*));
    hipMalloc((void**)&d_population, MAX_POPULATION_SIZE * sizeof(int*));
    hipMalloc((void**)&d_closest, MAX_POOL * sizeof(int*));

    // allocate memory for the data pointed to by the pointers on the device
    for (int i = 0; i < MAX_POPULATION_SIZE; i++) {
        hipMalloc((void**)&(d_population[i]), CHROMOSOMES_SIZE * sizeof(int));
    }

    for (int i = 0; i < MAX_POOL; i++) {
        hipMalloc((void**)&(d_closest[i]), CHROMOSOMES_SIZE * sizeof(int));
    }

    // copy the data from the host to the device
    hipMemcpy(d_original, original, CHROMOSOMES_SIZE * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < MAX_POPULATION_SIZE; i++) {
        hipMemcpy(d_population[i], population[i], CHROMOSOMES_SIZE * sizeof(int), hipMemcpyHostToDevice);
    }

    for (int i = 0; i < MAX_POOL; i++) {
        hipMemcpy(d_closest[i], closest[i], CHROMOSOMES_SIZE * sizeof(int), hipMemcpyHostToDevice);
    }

    auto now = std::chrono::system_clock::now();
    auto ms = std::chrono::time_point_cast<std::chrono::milliseconds>(now);
    auto start = ms.time_since_epoch().count();

    while (!check(original, population) && iterations < MAX_ITERATIONS)
    {
        iterations++;
        printf("Iteration: %d\n", iterations);

        for (int i = COMBINATIONAL_CROSSING_SIZE; i < MAX_POPULATION_SIZE; i++) {
            hipMemcpy(d_population[i], population[i], CHROMOSOMES_SIZE * sizeof(int), hipMemcpyHostToDevice);
        }

        get_distances_gpu << <MAX_POPULATION_SIZE, 256 >> > (d_population, d_original, d_distances);
        hipMemcpy(distances, d_distances, MAX_POPULATION_SIZE * sizeof(int), hipMemcpyHostToDevice);

        //CPU Implementation
        // get_distances(population, original, distances, sorted);

        get_closest_distances(population, closest, distances, sorted);

        combinational_crossing_gpu << <MAX_POOL, MAX_POOL >> > (d_population, d_closest);
        for (int i = 0; i < COMBINATIONAL_CROSSING_SIZE; i++) {
            hipMemcpy(population[i], d_population[i], CHROMOSOMES_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        }

        //CPU Implementation
        // combinational_crossing(population, closest);
        mutate(population, closest, COMBINATIONAL_CROSSING_SIZE);
    }

    now = std::chrono::system_clock::now();
    ms = std::chrono::time_point_cast<std::chrono::milliseconds>(now);
    auto end = ms.time_since_epoch().count();
    std::printf("Time taken: %2.4f s\n", (end - start) / 1000.0);

    delete distances;
    delete sorted;
    delete[] population;
    delete[] closest;

    return 0;
}